#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */


#include "hip/hip_runtime.h"
#include<iostream>

#include "hip/hip_texture_types.h"
 //#include "../source_code/common/book.h"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "../source_code/common/book.h"
#include "../source_code/common/cpu_anim.h"

#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED   0.25f

 // these exist on the GPU side
texture<float, 2>  texConstSrc;
texture<float, 2>  texIn;
texture<float, 2>  texOut;

__global__ void blend_kernel_2d(float* dst,
    bool dstOut) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float   t, l, c, r, b;
    if (dstOut) {
        t = tex2D(texIn, x, y - 1);
        l = tex2D(texIn, x - 1, y);
        c = tex2D(texIn, x, y);
        r = tex2D(texIn, x + 1, y);
        b = tex2D(texIn, x, y + 1);
    }
    else {
        t = tex2D(texOut, x, y - 1);
        l = tex2D(texOut, x - 1, y);
        c = tex2D(texOut, x, y);
        r = tex2D(texOut, x + 1, y);
        b = tex2D(texOut, x, y + 1);
    }
    dst[offset] = c + SPEED * (t + b + r + l - 4 * c);
}

__global__ void copy_const_kernel_2d(float* iptr) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float c = tex2D(texConstSrc, x, y);
    if (c != 0)
        iptr[offset] = c;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char* output_bitmap;
    float* dev_inSrc;
    float* dev_outSrc;
    float* dev_constSrc;
    CPUAnimBitmap* bitmap;

    hipEvent_t     start, stop;
    float           totalTime;
    float           frames;
};

void anim_gpu_2d(DataBlock* d, int ticks) {
    hipEventRecord(d->start, 0);
    dim3    blocks(DIM / 16, DIM / 16);
    dim3    threads(16, 16);
    CPUAnimBitmap* bitmap = d->bitmap;

    // since tex is global and bound, we have to use a flag to
    // select which is in/out per iteration
    volatile bool dstOut = true;
    for (int i = 0; i < 90; i++) {
        float* in, * out;
        if (dstOut) {
            in = d->dev_inSrc;
            out = d->dev_outSrc;
        }
        else {
            out = d->dev_inSrc;
            in = d->dev_outSrc;
        }
        copy_const_kernel_2d << <blocks, threads >> > (in);
        blend_kernel_2d << <blocks, threads >> > (out, dstOut);
        dstOut = !dstOut;
    }
    float_to_color << <blocks, threads >> > (d->output_bitmap,
        d->dev_inSrc);

    hipMemcpy(bitmap->get_ptr(),
        d->output_bitmap,
        bitmap->image_size(),
        hipMemcpyDeviceToHost);

    hipEventRecord(d->stop, 0);
    hipEventSynchronize(d->stop);
    float   elapsedTime;
    hipEventElapsedTime(&elapsedTime,
        d->start, d->stop);
    d->totalTime += elapsedTime;
    ++d->frames;
    printf("Average Time per frame:  %3.1f ms\n",
        d->totalTime / d->frames);
}

// clean up memory allocated on the GPU
void anim_exit(DataBlock* d) {
    hipUnbindTexture(texIn);
    hipUnbindTexture(texOut);
    hipUnbindTexture(texConstSrc);
    hipFree(d->dev_inSrc);
    hipFree(d->dev_outSrc);
    hipFree(d->dev_constSrc);

    hipEventDestroy(d->start);
    hipEventDestroy(d->stop);
}


int main2(void) {
    DataBlock   data;
    CPUAnimBitmap bitmap(DIM, DIM, &data);
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    hipEventCreate(&data.start);
    hipEventCreate(&data.stop);

    int imageSize = bitmap.image_size();

    hipMalloc((void**)&data.output_bitmap,
        imageSize);

    // assume float == 4 chars in size (ie rgba)
    hipMalloc((void**)&data.dev_inSrc,
        imageSize);
    hipMalloc((void**)&data.dev_outSrc,
        imageSize);
    hipMalloc((void**)&data.dev_constSrc,
        imageSize);

    hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    hipBindTexture2D(NULL, texConstSrc,
        data.dev_constSrc,
        desc, DIM, DIM,
        sizeof(float) * DIM);

    hipBindTexture2D(NULL, texIn,
        data.dev_inSrc,
        desc, DIM, DIM,
        sizeof(float) * DIM);

    hipBindTexture2D(NULL, texOut,
        data.dev_outSrc,
        desc, DIM, DIM,
        sizeof(float) * DIM);

    // initialize the constant data
    float* temp = (float*)malloc(imageSize);
    for (int i = 0; i < DIM * DIM; i++) {
        temp[i] = 0;
        int x = i % DIM;
        int y = i / DIM;
        if ((x > 300) && (x < 600) && (y > 310) && (y < 601))
            temp[i] = MAX_TEMP;
    }
    temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
    temp[DIM * 700 + 100] = MIN_TEMP;
    temp[DIM * 300 + 300] = MIN_TEMP;
    temp[DIM * 200 + 700] = MIN_TEMP;
    for (int y = 800; y < 900; y++) {
        for (int x = 400; x < 500; x++) {
            temp[x + y * DIM] = MIN_TEMP;
        }
    }
    hipMemcpy(data.dev_constSrc, temp,
        imageSize,
        hipMemcpyHostToDevice);

    // initialize the input data
    for (int y = 800; y < DIM; y++) {
        for (int x = 0; x < 200; x++) {
            temp[x + y * DIM] = MAX_TEMP;
        }
    }
    hipMemcpy(data.dev_inSrc, temp,
        imageSize,
        hipMemcpyHostToDevice);
    free(temp);

    bitmap.anim_and_exit((void (*)(void*, int))anim_gpu_2d,
        (void (*)(void*))anim_exit);
    return 0;
}

