#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include<iostream>

#include "hip/hip_texture_types.h"
//#include "../source_code/common/book.h"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "../source_code/common/book.h"
#include "../source_code/common/cpu_anim.h"

texture<int> testIn;
texture<int> testOUT;

__global__ void test_kernel(int* dst) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int c = tex1Dfetch(testIn, tid);
	dst[tid] = c * c;
}
__global__ void test_kernel_no_texture(int* dst, int * a) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int c = a[tid];
	dst[tid] = c * c;
}


int main(void) {
	int* a, *b;
	hipMalloc((void**)&a, sizeof(int) * 100000);
	hipMalloc((void**)&b, sizeof(int) * 100000);
	hipBindTexture(NULL, testIn, a, sizeof(int) * 100000);
	hipBindTexture(NULL, testOUT, b, sizeof(int) * 100000);
	
	int* temp = (int*)malloc(100000 * sizeof(int));
	for (int i = 0; i < 100000; i++) {
		temp[i] = i*i;
		//temp2[i] = 2 * i;
		//printf("temp = %d\n", temp[i]);
	}
	hipMemcpy(a, temp, sizeof(int) * 100000, hipMemcpyHostToDevice);
	//hipMemcpy(b, temp2, sizeof(int) * 100000, hipMemcpyHostToDevice);
	
	
	hipEvent_t     start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	test_kernel << <256, 256 >> > (b);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float   elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time to generate:  %3.1f ms\n", elapsedTime);

	hipEvent_t     start_2, stop_2;
	hipEventCreate(&start_2);
	hipEventCreate(&stop_2);
	hipEventRecord(start_2, 0);
	test_kernel_no_texture << <256, 256 >> > (b,a);
	hipEventRecord(stop_2, 0);
	hipEventSynchronize(stop_2);
	//float   elapsedTime;
	hipEventElapsedTime(&elapsedTime, start_2, stop_2);
	printf("Time to generate:  %3.1f ms\n", elapsedTime);




	int* output2 = (int*)malloc(100000 * sizeof(int));

	hipMemcpy(output2, b, 100000 * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < 100000; i++)
	{
		//printf("output2[%d]= %d\n", i, output2[i]);
	}
	hipUnbindTexture(testIn);
	hipUnbindTexture(testOUT);
	hipFree(a);
	hipFree(b);
	

	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(temp);

	return 0;
}