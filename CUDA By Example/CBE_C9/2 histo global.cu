
#include <hip/hip_runtime.h>
//#include "cuda.h"
//#include<iostream>
//
//#include "cuda_texture_types.h"
////#include "../source_code/common/book.h"
//#include "device_launch_parameters.h"
//#include "cuda.h"
//#include "cuda_runtime_api.h"
//#include "../source_code/common/book.h"
//#include "../source_code/common/cpu_anim.h"
//
//
//
//
//
//
//
//
//#define SIZE (100*1024*1024)
//
//__global__ void histo_kernel_global(unsigned char* buffer,
//	long size,
//	unsigned int* histo) {
//	int i = threadIdx.x + blockIdx.x * blockDim.x;
//	int stride = gridDim.x * blockDim.x;
//	while (i < size) {
//		histo[buffer[i]]++; //������Ǵ�ģ�������ԭ�Ӽӷ�
//		//atomicAdd(&histo[buffer[i]], 1);
//		i += stride;
//	}
//}
//int main(void) {
//	unsigned char* buffer =
//		(unsigned char*)big_random_block(SIZE);
//	// capture the start time
//   // starting the timer here so that we include the cost of
//   // all of the operations on the GPU.
//	cudaEvent_t     start, stop;
//	cudaEventCreate(&start);
//	cudaEventCreate(&stop);
//	cudaEventRecord(start, 0);
//
//
//	unsigned char *dev_buffer;
//	unsigned int *dev_histo;
//	cudaMalloc((void**)&dev_buffer, SIZE);
//	cudaMemcpy(dev_buffer, buffer, SIZE , cudaMemcpyHostToDevice);
//	cudaMalloc((void**)&dev_histo, 256 * sizeof(int));
//	cudaMemset(dev_histo, 0, 256 * sizeof(int));
//	cudaDeviceProp prop;
//	cudaGetDeviceProperties(&prop, 0);
//	int blocks = prop.multiProcessorCount;
//	histo_kernel_global << <blocks * 2, 256 >> > (dev_buffer, SIZE, dev_histo);
//
//	unsigned int histo[256];
//	cudaMemcpy(histo, dev_histo, 256 * sizeof(int), cudaMemcpyDeviceToHost);
//	cudaEventRecord(stop, 0);
//	float elapsedTime;
//	cudaEventElapsedTime(&elapsedTime, start, stop);
//	printf("Time to generate: %3.1f ms\n", elapsedTime);
//
//	for (int i = 0; i < SIZE; i++) {
//		histo[buffer[i]]--;
//	}
//	for (int i = 0; i < 256; i++) {
//		if (histo[i] != 0) {
//			printf("Failure at %d!  Off by %d\n", i, histo[i]);
//		}
//	}
//	cudaEventDestroy(start);
//	cudaEventDestroy(stop);
//	cudaFree(dev_histo);
//	cudaFree(dev_buffer);
//	free(buffer);
//	return 0;
//	
//	
//}//Time to generate: 162.2 ms 
//
