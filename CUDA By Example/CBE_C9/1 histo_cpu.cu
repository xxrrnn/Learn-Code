
#include <hip/hip_runtime.h>

//#include "../source_code/common/book.h"
//
//
//#define SIZE (100*1024*1024)
//
//int main(void) {
//	unsigned char* buffer = 
//			(unsigned char*)big_random_block(SIZE);
//	clock_t  start, stop;
//	start = clock();
//
//	unsigned int histo[256];
//	for (int i = 0; i < 256; i++)
//		histo[i] = 0;
//	for (int i = 0; i < SIZE; i++)
//		histo[buffer[i]]++;
//
//	stop = clock();
//	float elapsedTime = (float)(stop - start) / 
//						(float)CLOCKS_PER_SEC * 1000.0f;;
//	printf("Time to generate:  %3.1f ms\n", elapsedTime);
//
//	long histoCount = 0;
//	for (int i = 0; i < 256; i++) {
//		histoCount += histo[i];
//	}
//	printf("Histogram Sum:  %ld\n", histoCount);
//
//	free(buffer);
//	//system("pause");
//	return 0;
//}
///*Time to generate:  133.0 ms
//Histogram Sum:  104857600*/