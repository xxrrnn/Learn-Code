#include "hip/hip_runtime.h"
/*
�ִ�汾��������������:
prop���ֻ��ǲ�̫��д
cudamemcpy�㷴��Ŀ���Դ�����������豸�����豸ָ��ΪĿ�ꣻ
���豸��������������ָ��ΪĿ��
kernel�����������Ҫ��threadIdx.x��Ϊindex��������buffer[i]
�����ڿ�ʼ�����д��붼Ҫ���գ��� ��� �Լ�д debug �ο�ģʽ
*/

#include "hip/hip_runtime.h"
#include<iostream>

#include "hip/hip_texture_types.h"
//#include "../source_code/common/book.h"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "../source_code/common/book.h"
#include "../source_code/common/cpu_anim.h"

#define SIZE (100*1024*1024)

__global__ void histo_kernel_shared(unsigned char* buffer,
	long size,
	unsigned int* histo) {
	
	__shared__ unsigned int temp[256];
	temp[threadIdx.x] = 0;
	__syncthreads();
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	while (i < size) {
		atomicAdd( &temp[buffer[i]], 1 );
		i += stride;
	}
	__syncthreads();
	atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}

int main(void) {
	unsigned char* buffer =
		(unsigned char*)big_random_block(SIZE);
	unsigned char* dev_buffer;
	unsigned int* dev_histo;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipMalloc((void**)&dev_buffer, SIZE);
	hipMalloc((void**)&dev_histo, 256 * sizeof(int));
	hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice);
	hipMemset(dev_histo, 0, 256 * sizeof(int));

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int blocks = prop.multiProcessorCount;

	histo_kernel_shared << <blocks * 2, 256 >> > (dev_buffer, SIZE, dev_histo);
	
	unsigned int histo[256];
	hipMemcpy(histo, dev_histo, 256 * sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time to generate: %3.1f ms\n", elapsedTime);

	long histoCount = 0;
	for (int i = 0; i < 256; i++) {
		histoCount += histo[i];
	}
	printf("Histogram sum: %ld \n", histoCount);

	for (int i = 0; i < SIZE; i++) {
		histo[buffer[i]]--;
	}
	for (int i = 0; i < 256; i++) {
		if (histo[i] != 0) {
			printf("error: %d \n", i);
		}
	}
	hipFree(dev_buffer);
	hipFree(dev_histo);
	free(buffer);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}

/*Time to generate: 44.8 ms
Histogram sum: 104857600*/