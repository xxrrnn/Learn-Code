#include<iostream>

#include "hip/hip_texture_types.h"
//#include "../source_code/common/book.h"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "../source_code/common/book.h"
#define SIZE (64*1024*1024)

float copy_test_hostAlloc(int size, bool up) {
	int* a, * dev_a;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipHostAlloc((void**)&a,size*sizeof(*a),hipHostMallocDefault);
	hipMalloc((void**)&dev_a, size * sizeof(*dev_a));
	//memset(a, 0, size * sizeof(*a));
	//hipMemset(dev_a, 0, size * sizeof(*a));
	hipEventRecord(start, 0);
	for (int i = 0; i < 100; i++) {
		if (up) { //upָ�������豸
			hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice);
		}
		else {
			hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost);
		}
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop); //�����������
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);


	hipFree(dev_a);
	hipHostFree(a);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return elapsedTime;
}
float copy_test_malloc(int size, bool up) {
	int* a, * dev_a;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	a = (int*)malloc(size * sizeof(*a));
	hipMalloc((void**)&dev_a, size * sizeof(*dev_a));
	//memset(a, 0, size * sizeof(*a));
	//hipMemset(dev_a, 0, size * sizeof(*a));
	hipEventRecord(start, 0);
	for (int i = 0; i < 100; i++) {
		if (up) { //upָ�������豸
			hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice);
		}
		else {
			hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost);
		}
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop); //�����������
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);


	hipFree(dev_a);
	free(a);
	hipEventDestroy(start);
	hipEventDestroy(stop);


	return elapsedTime;
}

int main1(void) {
	float MB = (float)100 * SIZE * sizeof(int) / 1024 / 1024;
	float elapsedTime;
	elapsedTime = copy_test_malloc(SIZE, true);
	printf("Time using hipMalloc: %3.If ms\n",
		elapsedTime);
	printf("\tMB / s during copy up : % 3.1f\n" ,
		MB / (elapsedTime / 1000));
	elapsedTime = copy_test_malloc(SIZE, false);
	printf("Time using hipMalloc: %3.If ms\n",
		elapsedTime);
	printf("\tMB / s during copy down : % 3.1f\n",
		MB / (elapsedTime / 1000));

	elapsedTime = copy_test_hostAlloc(SIZE, true);
	printf("Time using hipHostAlloc: %3.If ms\n",
		elapsedTime);
	printf("\tMB / s during copy up : % 3.1f\n",
		MB / (elapsedTime / 1000));
	elapsedTime = copy_test_hostAlloc(SIZE, false);
	printf("Time using hipHostAlloc: %3.If ms\n",
		elapsedTime);
	printf("\tMB / s during copy down : % 3.1f\n",
		MB / (elapsedTime / 1000));

	return 0;
}
/*
Time using hipMalloc: 4536 ms
		MB / s during copy up :  5644.1
Time using hipMalloc: 4580 ms
		MB / s during copy down :  5590.1
Time using hipHostAlloc: 4357 ms
		MB / s during copy up :  5875.9
Time using hipHostAlloc: 4400 ms
		MB / s during copy down :  5818.6
		*/