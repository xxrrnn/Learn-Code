#include "hip/hip_runtime.h"
#include<iostream>

#include "hip/hip_texture_types.h"
//#include "../source_code/common/book.h"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
//#include "../source_code/common/book.h"

#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)
//#define FULL_DATA_SIZE   (N*200)

__global__ void single_stream_kernel(int* a, int* b, int* c) { //�����⺬�壬ֻ��һ����ͨ����
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < N) {
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}

int main(void) {
	hipDeviceProp_t prop;
	int whichdevice;
	hipGetDevice(&whichdevice);
	hipGetDeviceProperties(&prop, whichdevice);
	if (!prop.deviceOverlap) {
		printf("device will not handle overlaps,so no speed up from streams\n");
		return 0;
	}


	int* host_a, * host_b, * host_c;
	int* dev_a, * dev_b, * dev_c;
	float elapsedTime;
	hipStream_t stream0;
	hipStreamCreate(&stream0);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));

	for (int i = 0; i < FULL_DATA_SIZE; i++) {
		host_a[i] = rand();
		host_b[i] = rand();
	}

	hipEventRecord(start, 0);
	for (int i = 0; i < FULL_DATA_SIZE; i += N) {
		hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice,stream0);
		hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		single_stream_kernel << <N / 256, 256 ,0,stream0>> > (dev_a, dev_b, dev_c);
		hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToDevice, stream0);

	}
	
	hipStreamSynchronize(stream0);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time taken: %3.1f ms\n", elapsedTime);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipStreamDestroy(stream0);

	return 0;
}
/*
Time taken: 50.8 ms

Time taken: 756.0 ms
*/