#include "hip/hip_runtime.h"
#include<iostream>

#include "hip/hip_texture_types.h"
//#include "../source_code/common/book.h"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
//#include "../source_code/common/book.h"

#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)
//#define FULL_DATA_SIZE   (N*200)

__global__ void double_stream_kernel(int* a, int* b, int* c) { //�����⺬�壬ֻ��һ����ͨ����
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < N) {
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}

int main(void) {
	hipDeviceProp_t prop;
	int whichdevice;
	hipGetDevice(&whichdevice);
	hipGetDeviceProperties(&prop, whichdevice);
	if (!prop.deviceOverlap) {
		printf("device will not handle overlaps,so no speed up from streams\n");
		return 0;
	}


	int* host_a, * host_b, * host_c;
	int* dev_a, * dev_b, * dev_c;
	int* dev_a1, * dev_b1, * dev_c1;
	float elapsedTime;
	hipStream_t stream0,stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));
	hipMalloc((void**)&dev_a1, N * sizeof(int));
	hipMalloc((void**)&dev_b1, N * sizeof(int));
	hipMalloc((void**)&dev_c1, N * sizeof(int));

	for (int i = 0; i < FULL_DATA_SIZE; i++) {
		host_a[i] = rand();
		host_b[i] = rand();
	}
	hipEventRecord(start, 0);
	for(int i = 0; i < FULL_DATA_SIZE; i += 2 * N) {
		hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice,stream0);
		hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice,stream0);
		hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice,stream1);
		double_stream_kernel << <N / 256, 256, 0, stream0 >> > (dev_a, dev_b, dev_c);
		double_stream_kernel << <N / 256, 256, 0, stream1 >> > (dev_a1, dev_b1, dev_c1);

		hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
	}
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time taken:  %3.1f ms\n", elapsedTime);




	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_a1);
	hipFree(dev_b1);
	hipFree(dev_c1);
	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);
	return 0;
}
/*
Time taken:  44.0 ms ����һЩ

Time taken:  633.3 ms
*/