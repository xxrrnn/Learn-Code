#include<iostream>
//#include "../source_code/common/book.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h" 

#define N 100
__global__ void add(int* a, int* b, int* c) {
	int tid = threadIdx.x;  //�����������������
	if (tid < N) {
		c[tid] = a[tid] + b[tid];
	}
}


int main1(void) {
	int a[N], b[N], c[N];
	int* dev_a, * dev_b, * dev_c;
	//��gpu�Ϸ����ڴ�
	hipMalloc((void**)&dev_a, N*sizeof(int));
	hipMalloc((void**)&dev_b, N*sizeof(int));
	hipMalloc((void**)&dev_c, N*sizeof(int));
	//��cpu��Ϊ����ab��ֵ
	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}
	//��ab���Ƶ�gpu
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	add <<<1,N>>> (dev_a, dev_b, dev_c); //N���豸��ִ�к˺���ʱʹ�õĲ����߳̿������
	hipMemcpy(c, dev_c, N * sizeof(int),hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}