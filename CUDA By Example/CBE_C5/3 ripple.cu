#include "hip/hip_runtime.h"
#include<iostream>
//#include "../source_code/common/book.h"
#include ""
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h" 
#include "../source_code/common/cpu_anim.h"
#define DIM 2048
#define PI 3.1415926535897932f

__global__ void kernel_ripple(unsigned char* ptr, int ticks) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // now calculate the value at that position
    float fx = x - DIM / 2;
    float fy = y - DIM / 2;
    float d = sqrtf(fx * fx + fy * fy);
    unsigned char grey = (unsigned char)(128.0f + 127.0f *
        cos(d / 10.0f - ticks / 7.0f) /
        (d / 10.0f + 1.0f));
    ptr[offset * 4 + 0] = grey;
    ptr[offset * 4 + 1] = grey;
    ptr[offset * 4 + 2] = grey;
    ptr[offset * 4 + 3] = 255;
}


struct DataBlock {
	unsigned char* dev_bitmap;
	CPUAnimBitmap* bitmap;
};

void cleanup(DataBlock* d) {
	hipFree(d->dev_bitmap);
}

void generate_frame(DataBlock* d, int ticks) {
	dim3 blocks(DIM / 16, DIM / 16);
	dim3 threads(16, 16);
    kernel_ripple << <blocks, threads >> > (d->dev_bitmap, ticks);
	hipMemcpy(d->bitmap->get_ptr(), d->dev_bitmap, d->bitmap->image_size(), hipMemcpyDeviceToHost);
}

int main3(void) {
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	hipMalloc((void**)&data.dev_bitmap, bitmap.image_size());
	bitmap.anim_and_exit((void(*)(void*, int))generate_frame,
		(void(*)(void*))cleanup);
	return 0;
}