
#include <hip/hip_runtime.h>
﻿//#include<iostream>
////#include "../source_code/common/book.h"
//#include "device_launch_parameters.h"
//#include "cuda.h"
//#include "cuda_runtime_api.h" 
//
//#define N 100
//__global__ void add(int* a, int* b, int* c) {
//	int tid = blockIdx.x;  //计算该索引处的数据
//	if (tid < N) {
//		c[tid] = a[tid] + b[tid];
//	}
//}
//
//
//int main01(void) {
//	int a[N], b[N], c[N];
//	int* dev_a, * dev_b, * dev_c;
//	//在gpu上分配内存
//	cudaMalloc((void**)&dev_a, N*sizeof(int));
//	cudaMalloc((void**)&dev_b, N*sizeof(int));
//	cudaMalloc((void**)&dev_c, N*sizeof(int));
//	//在cpu上为数组ab赋值
//	for (int i = 0; i < N; i++) {
//		a[i] = -i;
//		b[i] = i * i;
//	}
//	//将ab复制到gpu
//	cudaMemcpy(dev_a, a, N * sizeof(int), cudaMemcpyHostToDevice);
//	cudaMemcpy(dev_b, b, N * sizeof(int), cudaMemcpyHostToDevice);
//
//	add <<<N, 1>>> (dev_a, dev_b, dev_c); //N是设备在执行核函数时使用的并行线程块的数量
//	cudaMemcpy(c, dev_c, N * sizeof(int),cudaMemcpyDeviceToHost);
//	for (int i = 0; i < N; i++) {
//		printf("%d + %d = %d\n", a[i], b[i], c[i]);
//	}
//
//	cudaFree(dev_a);
//	cudaFree(dev_b);
//	cudaFree(dev_c);
//
//	return 0;
//}