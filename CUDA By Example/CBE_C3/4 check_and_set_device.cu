#include<iostream>
//#include "../source_code/common/book.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

int main(void) {
	hipDeviceProp_t prop;
	int dev;

	hipGetDevice(&dev);
	printf("ID of current CUDA device: %d \n", dev);

	memset(&prop, 0, sizeof(hipDeviceProp_t));

	prop.major = 1;
	prop.minor = 3;
	hipChooseDevice(&dev, &prop);
	printf("ID of CUDA device closest to revision 1.3: %d/n", dev);
	hipSetDevice(dev);
}